#include "hip/hip_runtime.h"
/**
 * FILE: lake.cu
 * DESCRIPTION: Original code for the lake calculation with the addition
 *   of the 9pt calculation. It also uses the new CUDA gpu call to calculate
 *   the change as well. In addition it uses MPI to spread the calcuation across
 *   multiple nodes.
 * AUTHORS:
 *   tmleibe2 Trevor M Leibert
 *   bjlayko Benjamin J Layko
 *   pjhamb  Palash Jhamb
 */
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#include "mpi.h"

#define _USE_MATH_DEFINES

#define XMIN 0.0
#define XMAX 1.0
#define YMIN 0.0
#define YMAX 1.0

#define MAX_PSZ 10
#define TSCALE 1.0
#define VSQR 0.1
#define ROOT 0
#define FILENAME_LEN 50
#define DEBUG(message) fprintf(stderr, message " proc %d, line %d file %s\n", rank, __LINE__, __FILE__)

void init(double *u, double *pebbles, int nrows, int ncols);
void evolve(double *un, double *uc, double *uo, double *pebbles, int nrows, int ncols, double h, double dt, double t);
void evolve9pt(double *un, double *uc, double *uo, double *pebbles, int nrows, int ncols, double h, double dt, double t);
int tpdt(double *t, double dt, double end_time);
void print_heatmap(const char *filename, double *u, int nrows, int ncols, double h, int y_offset);
void init_pebbles(double *p, int pn, int n);

void run_cpu(double *u, double *u0, double *u1, double *pebbles, int nrows, int ncols, double h, double end_time);

extern void run_gpu(double *u, double *u0, double *u1, double *pebbles, int nrows, int ncols, double h, double end_time, int nthreads);

int main(int argc, char *argv[])
{

  if (argc != 5)
  {
    printf("Usage: %s npoints npebs time_finish nthreads \n", argv[0]);
    return 1;
  }

  /* MPI variables and init */
  int rank, numproc;
  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &numproc);

  int npoints = atoi(argv[1]);

  // process-specific dimension information
  int ncols = npoints;
  // rows are split evenly across processes, with an extra row on the top and bottom for values
  // to be received from other processes
  int nrows = npoints / numproc;
  // allocate remainder of the split across first n processes
  if (rank < npoints % numproc)
  {
    nrows += 1;
  }

  // set up array of how rows each thread is responsible for
  // and how many total entries a thread is responsible for
  // and displacements into the source array for mpi_gatherv
  int *row_counts = (int *)malloc(sizeof(int) * numproc);
  int *total_counts = (int *)malloc(sizeof(int) * numproc);
  int *displacements = (int *)malloc(sizeof(int) * numproc);
  int running_sum = 0;
  for (int i = 0; i < numproc; i++)
  {
    row_counts[i] = npoints / numproc;
    if (i < npoints % numproc)
    {
      row_counts[i] += 1;
    }
    total_counts[i] = row_counts[i] * ncols;
    displacements[i] = running_sum;
    running_sum += total_counts[i];
  }

  int npebs = atoi(argv[2]);
  double end_time = (double)atof(argv[3]);
  int nthreads = atoi(argv[4]);

  // square area of the total grid
  int narea = npoints * npoints;

  double *u_i0, *u_i1;
  double *u_cpu, *u_gpu, *pebs_local;
  double h;

  double elapsed_cpu, elapsed_gpu;
  struct timeval cpu_start, cpu_end, gpu_start, gpu_end;

  u_i0 = (double *)malloc(sizeof(double) * (nrows + 2) * ncols);
  u_i1 = (double *)malloc(sizeof(double) * (nrows + 2) * ncols);

  // all other arrays use the new smaller grid sizes, but pebbles need to
  // be thrown over the entire original array size
  // we'll be copying the pebbles over to each smaller array, called pebs_local
  pebs_local = (double *)malloc(sizeof(double) * ncols * nrows);

  u_cpu = (double *)malloc(sizeof(double) * nrows * ncols);
  u_gpu = (double *)malloc(sizeof(double) * nrows * ncols);

  printf("Running %s with (%d x %d) grid, until %f, with %d threads\n", argv[0], nrows, ncols, end_time, nthreads);

  h = (XMAX - XMIN) / npoints;

  /* only initialzie pebbles in the root so we all share the same info */
  double *pebs = NULL;
  if (rank == ROOT)
  {
    pebs = (double *)malloc(sizeof(double) * narea);
    if (pebs == NULL)
    {
      fprintf(stderr, "Couldn't allocate pebbles\n");
      exit(1);
    }
    init_pebbles(pebs, npebs, npoints);
  }

  MPI_Scatterv(pebs, total_counts, displacements, MPI_DOUBLE, pebs_local, nrows * ncols, MPI_DOUBLE, ROOT, MPI_COMM_WORLD);
  if (rank == ROOT)
  {
    free(pebs);
  }

  init(u_i0 + ncols, pebs_local, nrows, ncols);
  init(u_i1 + ncols, pebs_local, nrows, ncols);

  // print one file per node
  char filename[FILENAME_LEN + 1];
  snprintf(filename, FILENAME_LEN, "lake_i_%d.dat", rank);
  print_heatmap(filename, u_i0 + ncols, nrows, ncols, h, displacements[rank] / ncols);

  gettimeofday(&cpu_start, NULL);
  run_cpu(u_cpu, u_i0, u_i1, pebs_local, nrows, ncols, h, end_time);
  gettimeofday(&cpu_end, NULL);

  elapsed_cpu = ((cpu_end.tv_sec + cpu_end.tv_usec * 1e-6) - (cpu_start.tv_sec + cpu_start.tv_usec * 1e-6));
  printf("Node %d: CPU took %f seconds\n", rank, elapsed_cpu);

  gettimeofday(&gpu_start, NULL);
  // TODO uncomment this
  run_gpu(u_gpu, u_i0, u_i1, pebs_local, nrows, ncols, h, end_time, nthreads);
  gettimeofday(&gpu_end, NULL);
  elapsed_gpu = ((gpu_end.tv_sec + gpu_end.tv_usec * 1e-6) - (gpu_start.tv_sec + gpu_start.tv_usec * 1e-6));
  printf("Node %d: GPU took %f seconds\n", rank, elapsed_gpu);

  // generate file name by node id
  // TODO change filenames
  snprintf(filename, FILENAME_LEN, "lake_f_cpu_%d.dat", rank);
  print_heatmap(filename, u_cpu, nrows, ncols, h, displacements[rank] / ncols);
  snprintf(filename, FILENAME_LEN, "lake_f_gpu_%d.dat", rank);
  print_heatmap(filename, u_gpu, nrows, ncols, h, displacements[rank] / ncols);

  free(u_i0);
  free(u_i1);
  free(pebs_local);
  free(u_cpu);
  free(u_gpu);
  free(row_counts);
  free(total_counts);
  free(displacements);

  // teardown mpi
  MPI_Finalize();

  return 0;
}

void run_cpu(double *u, double *u0, double *u1, double *pebbles, int nrows, int ncols, double h, double end_time)
{
  int rank, numproc;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &numproc);

  // determine our partners for send/receive
  int back_partner = (rank - 1) < 0 ? numproc - 1 : rank - 1;
  int forward_partner = (rank + 1) >= numproc ? 0 : rank + 1;

  double *un, *uc, *uo;
  double t, dt;

  un = (double *)malloc(sizeof(double) * nrows * ncols);
  // need 2 extra rows for the arrays that will be communicated back and forth
  uc = (double *)malloc(sizeof(double) * (nrows + 2) * ncols);
  uo = (double *)malloc(sizeof(double) * (nrows + 2) * ncols);
  if (un == NULL || uc == NULL || uo == NULL)
  {
    fprintf(stderr, "Error allocating run arrays\n");
    exit(1);
  }

  // copy initialized parts of history matrices into their local counterparts
  memcpy(uo + ncols, u0 + ncols, sizeof(double) * nrows * ncols);
  memcpy(uc + ncols, u1 + ncols, sizeof(double) * nrows * ncols);

  t = 0.;
  dt = h / 2.;

  while (1)
  {
    // synchronize with mpi before running evolve

    // array to hold send/receive requests for both arrays
    MPI_Request all_requests[8];
    // queue up non-blocking receives
    MPI_Irecv(uo, ncols, MPI_DOUBLE, back_partner, 0, MPI_COMM_WORLD, all_requests);
    MPI_Irecv(uc, ncols, MPI_DOUBLE, back_partner, 1, MPI_COMM_WORLD, all_requests + 1);
    MPI_Irecv(uo + (nrows + 1) * ncols, ncols, MPI_DOUBLE, forward_partner, 2, MPI_COMM_WORLD, all_requests + 2);
    MPI_Irecv(uc + (nrows + 1) * ncols, ncols, MPI_DOUBLE, forward_partner, 3, MPI_COMM_WORLD, all_requests + 3);
    // queue up non-blocking sends
    MPI_Isend(uc + ncols, ncols, MPI_DOUBLE, back_partner, 3, MPI_COMM_WORLD, all_requests + 4);
    MPI_Isend(uo + ncols, ncols, MPI_DOUBLE, back_partner, 2, MPI_COMM_WORLD, all_requests + 5);
    MPI_Isend(uc + nrows * ncols, ncols, MPI_DOUBLE, forward_partner, 1, MPI_COMM_WORLD, all_requests + 6);
    MPI_Isend(uo + nrows * ncols, ncols, MPI_DOUBLE, forward_partner, 0, MPI_COMM_WORLD, all_requests + 7);

    // wait on all requests
    MPI_Waitall(8, all_requests, MPI_STATUS_IGNORE);

    // evolve the simulation
    // offset into the "real" data inside of uc and uo
    // first and last rows are the "extra" ones
    evolve9pt(un, uc + ncols, uo + ncols, pebbles, nrows, ncols, h, dt, t);

    memcpy(uo + ncols, uc + ncols, sizeof(double) * nrows * ncols);
    memcpy(uc + ncols, un, sizeof(double) * nrows * ncols);

    if (!tpdt(&t, dt, end_time))
      break;
  }

  memcpy(u, un, sizeof(double) * nrows * ncols);
}

void init_pebbles(double *p, int pn, int n)
{
  int i, j, k, idx;
  int sz;

  srand(time(NULL));
  memset(p, 0, sizeof(double) * n * n);

  for (k = 0; k < pn; k++)
  {
    i = rand() % (n - 4) + 2;
    j = rand() % (n - 4) + 2;
    sz = rand() % MAX_PSZ;
    // transpose placements
    idx = i + j * n;
    p[idx] = (double)sz;
  }
}

double f(double p, double t)
{
  return -expf(-TSCALE * t) * p;
}

int tpdt(double *t, double dt, double tf)
{
  if ((*t) + dt > tf)
    return 0;
  (*t) = (*t) + dt;
  return 1;
}

void init(double *u, double *pebbles, int nrows, int ncols)
{
  int i, j, idx;

  for (i = 0; i < nrows; i++)
  {
    for (j = 0; j < ncols; j++)
    {
      idx = j + i * ncols;
      u[idx] = f(pebbles[idx], 0.0);
    }
  }
}

void evolve(double *un, double *uc, double *uo, double *pebbles, int nrows, int ncols, double h, double dt, double t)
{
  int i, j, idx;

  for (i = 0; i < nrows; i++)
  {
    for (j = 0; j < ncols; j++)
    {
      idx = j + i * ncols;

      if (j == 0 || j == ncols - 1)
      {
        un[idx] = 0.;
      }
      else
      {
        un[idx] = 2 * uc[idx] - uo[idx] + VSQR * (dt * dt) * ((uc[idx - 1] + uc[idx + 1] + uc[idx + ncols] + uc[idx - ncols] - 4 * uc[idx]) / (h * h) + f(pebbles[idx], t));
      }
    }
  }
}

/**
 * @brief Version of evolve usinga 9-point stencil to calculate the difference
 *
 * @param un m by n results matrix
 * @param uc m + 2 by n input history matrix
 * @param uo m + 2 by n input history matrix
 * @param pebbles m by n input pebble matrix
 * @param nrows number of rows
 * @param ncols number of cols
 * @param h height of the overall matrix
 * @param dt time step
 * @param t current time
 */
void evolve9pt(double *un, double *uc, double *uo, double *pebbles, int nrows, int ncols, double h, double dt, double t)
{
  int i, j, idx;
  double sum = 0.0;
  for (i = 0; i < nrows; i++)
  {
    for (j = 0; j < ncols; j++)
    {
      idx = j + i * ncols;

      if (j == 0 || j == ncols - 1)
      {
        un[idx] = 0.;
      }
      else
      {
        // 9 point pseudocode adapted to C
        un[idx] = 2 * uc[idx] - uo[idx] + VSQR * (dt * dt) * ((uc[idx - 1] + uc[idx + 1] + uc[idx + ncols] + uc[idx - ncols] + ((uc[idx - ncols - 1] + uc[idx - ncols + 1] + uc[idx + ncols - 1] + uc[idx + ncols + 1]) / 4) - 5 * uc[idx]) / (h * h) + f(pebbles[idx], t));
      }
      sum += un[idx];
    }
  }
}

void print_heatmap(const char *filename, double *u, int nrows, int ncols, double h, int y_offset)
{
  int i, j, idx;

  FILE *fp = fopen(filename, "w");

  for (i = 0; i < nrows; i++)
  {
    for (j = 0; j < ncols; j++)
    {
      idx = j + i * ncols;
      // transpose print cols
      fprintf(fp, "%f %f %f\n", j * h, (i + y_offset) * h, u[idx]);
    }
  }

  fclose(fp);
}
