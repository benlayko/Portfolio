#include "hip/hip_runtime.h"
/**
 * FILE: lakegpu_mpi.cu
 * DESCRIPTION: Implements the lake surface calculation
 *   using CUDA code and MPI so that it can be spread
 *   across multiple nodes.
 * AUTHORS:
 *   tmleibe2 Trevor M Leibert
 *   bjlayko Benjamin J Layko
 *   pjhamb  Palash Jhamb
 */
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "mpi.h"

#define __DEBUG

#define XMIN 0.0
#define XMAX 1.0
#define YMIN 0.0
#define YMAX 1.0

#define MAX_PSZ 10
#define TSCALE 1.0
#define VSQR 0.1

#define CUDA_CALL(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CUDA_CHK_ERR() __cudaCheckError(__FILE__, __LINE__)

/**************************************
 * void __cudaSafeCall(hipError_t err, const char *file, const int line)
 * void __cudaCheckError(const char *file, const int line)
 *
 * These routines were taken from the GPU Computing SDK
 * (http://developer.nvidia.com/gpu-computing-sdk) include file "cutil.h"
 **************************************/
inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef __DEBUG

#pragma warning(push)
#pragma warning(disable : 4127) // Prevent warning on do-while(0);
  do
  {
    if (hipSuccess != err)
    {
      fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
              file, line, hipGetErrorString(err));
      exit(-1);
    }
  } while (0);
#pragma warning(pop)
#endif // __DEBUG
  return;
}

inline void __cudaCheckError(const char *file, const int line)
{
#ifdef __DEBUG
#pragma warning(push)
#pragma warning(disable : 4127) // Prevent warning on do-while(0);
  do
  {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
      fprintf(stderr, "cudaCheckError() failed at %s:%i : %s.\n",
              file, line, hipGetErrorString(err));
      exit(-1);
    }
    // More careful checking. However, this will affect performance.
    // Comment if not needed.
    /*err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }*/
  } while (0);
#pragma warning(pop)
#endif // __DEBUG
  return;
}

static __device__ double f(double p, double t)
{
  return -expf(-TSCALE * t) * p;
}

__global__ void EvolveKernel(double *un, double *uc, double *uo, double *pebbles, int nrows, int ncols, double h, double t, double dt)
{
  // determine gpu thread's index into the arrays
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // gate off gpu from illegal memory accesses
  if (x > ncols - 1 || y > nrows - 1)
  {
    return;
  }

  int idx = y * ncols + x;

  // set edges to 0
  if (x == 0 || x == ncols - 1)
  {
    un[idx] = 0.;
  }
  else
  {
    un[idx] = 2 * uc[idx] - uo[idx] + VSQR * (dt * dt) * ((uc[idx - 1] + uc[idx + 1] + uc[idx + ncols] + uc[idx - ncols] + ((uc[idx - ncols - 1] + uc[idx - ncols + 1] + uc[idx + ncols - 1] + uc[idx + ncols + 1]) / 4) - 5 * uc[idx]) / (h * h) + f(pebbles[idx], t));
  }
}

static int tpdt(double *t, double dt, double tf)
{
  if ((*t) + dt > tf)
    return 0;
  (*t) = (*t) + dt;
  return 1;
}

void run_gpu(double *u, double *u0, double *u1, double *pebbles, int nrows, int ncols, double h, double end_time, int nthreads)
{
  hipEvent_t kstart, kstop;
  float ktime;

  /* HW2: Define your local variables here */

  /* Determine rank and number of processes */
  int rank, numproc;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &numproc);

  // determine our partners for send/receive
  int back_partner = (rank - 1) < 0 ? numproc - 1 : rank - 1;
  int forward_partner = (rank + 1) >= numproc ? 0 : rank + 1;

  /* device arrays and timing vars - need to add host arrays for mpi */
  double *uc, *uo, *un_d, *uc_d, *uo_d, *pebbles_d, t, dt;

  t = 0.;
  dt = h / 2.;
  /* number of blocks to launch with */
  int nblocks = ncols / nthreads;

  dim3 dimblocks(nblocks, nblocks);
  dim3 dimgrid(nthreads, nthreads);

  /* Set up device timers */
  CUDA_CALL(hipSetDevice(0));
  CUDA_CALL(hipEventCreate(&kstart));
  CUDA_CALL(hipEventCreate(&kstop));

  /* HW2: Add CUDA kernel call preperation code here */
  /* allocate device arrays */
  CUDA_CALL(hipMalloc(&un_d, sizeof(double) * nrows * ncols));
  CUDA_CALL(hipMalloc(&uc_d, sizeof(double) * (nrows + 2) * ncols));
  CUDA_CALL(hipMalloc(&uo_d, sizeof(double) * (nrows + 2) * ncols));
  CUDA_CALL(hipMalloc(&pebbles_d, sizeof(double) * nrows * ncols));

  /* allocate host arrays for mpi */
  uc = (double *)malloc(sizeof(double) * (nrows + 2) * ncols);
  uo = (double *)malloc(sizeof(double) * (nrows + 2) * ncols);
  if (uc == NULL || uo == NULL)
  {
    fprintf(stderr, "Error allocating host mpi arrays\n");
    exit(1);
  }

  /* copy in initial state into host arrays */
  memcpy(uo + ncols, u0 + ncols, sizeof(double) * nrows * ncols);
  memcpy(uc + ncols, u1 + ncols, sizeof(double) * nrows * ncols);

  /* Start GPU computation timer */
  CUDA_CALL(hipEventRecord(kstart, 0));

  /* copy in initial state into device arrays */
  CUDA_CALL(hipMemcpy(pebbles_d, pebbles, sizeof(double) * nrows * ncols, hipMemcpyHostToDevice));

  /* HW2: Add main lake simulation loop here */

  while (1)
  {
    // synchronize with mpi before running evolve
    // array to hold send/receive requests for both arrays
    MPI_Request all_requests[8];
    // queue up non-blocking receives
    MPI_Irecv(uo, ncols, MPI_DOUBLE, back_partner, 0, MPI_COMM_WORLD, all_requests);
    MPI_Irecv(uc, ncols, MPI_DOUBLE, back_partner, 1, MPI_COMM_WORLD, all_requests + 1);
    MPI_Irecv(uo + (nrows + 1) * ncols, ncols, MPI_DOUBLE, forward_partner, 2, MPI_COMM_WORLD, all_requests + 2);
    MPI_Irecv(uc + (nrows + 1) * ncols, ncols, MPI_DOUBLE, forward_partner, 3, MPI_COMM_WORLD, all_requests + 3);
    // queue up non-blocking sends
    MPI_Isend(uc + ncols, ncols, MPI_DOUBLE, back_partner, 3, MPI_COMM_WORLD, all_requests + 4);
    MPI_Isend(uo + ncols, ncols, MPI_DOUBLE, back_partner, 2, MPI_COMM_WORLD, all_requests + 5);
    MPI_Isend(uc + nrows * ncols, ncols, MPI_DOUBLE, forward_partner, 1, MPI_COMM_WORLD, all_requests + 6);
    MPI_Isend(uo + nrows * ncols, ncols, MPI_DOUBLE, forward_partner, 0, MPI_COMM_WORLD, all_requests + 7);

    // wait on all requests
    MPI_Waitall(8, all_requests, MPI_STATUS_IGNORE);

    // copy freshly synced mpi arrays into the gpu
    CUDA_CALL(hipMemcpy(uo_d, uo, sizeof(double) * (nrows + 2) * ncols, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(uc_d, uc, sizeof(double) * (nrows + 2) * ncols, hipMemcpyHostToDevice));

    EvolveKernel<<<dimblocks, dimgrid>>>(un_d, uc_d + ncols, uo_d + ncols, pebbles_d, nrows, ncols, h, t, dt);

    // copy updated arrays back to the mpi holding buffers to be synced - updating orders
    // at the same time

    // offset here since both source and dest have extra rows
    CUDA_CALL(hipMemcpy(uo + ncols, uc_d + ncols, sizeof(double) * nrows * ncols, hipMemcpyDeviceToHost));
    // uc doesn't have extra rows, so no offset here
    CUDA_CALL(hipMemcpy(uc + ncols, un_d, sizeof(double) * nrows * ncols, hipMemcpyDeviceToHost));

    if (!tpdt(&t, dt, end_time))
      break;
  }

  /* copy out un device array after last timestep */
  CUDA_CALL(hipMemcpy(u, un_d, sizeof(double) * nrows * ncols, hipMemcpyDeviceToHost));

  /* Stop GPU computation timer */
  CUDA_CALL(hipEventRecord(kstop, 0));
  CUDA_CALL(hipEventSynchronize(kstop));
  CUDA_CALL(hipEventElapsedTime(&ktime, kstart, kstop));
  printf("GPU computation: %f msec\n", ktime);

  /* HW2: Add post CUDA kernel call processing and cleanup here */
  CUDA_CALL(hipFree(un_d));
  CUDA_CALL(hipFree(uc_d));
  CUDA_CALL(hipFree(uo_d));
  CUDA_CALL(hipFree(pebbles_d));

  /* timer cleanup */
  CUDA_CALL(hipEventDestroy(kstart));
  CUDA_CALL(hipEventDestroy(kstop));
}
