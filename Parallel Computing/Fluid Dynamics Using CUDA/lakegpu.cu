/**
 * FILE: lakegpu.cu
 * DESCRIPTION: Implements the lake surface calculation
 *   using CUDA code.
 * AUTHORS:
 *   tmleibe2 Trevor M Leibert
 *   bjlayko Benjamin J Layko
 *   pjhamb  Palash Jhamb
 */
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define __DEBUG

#define XMIN 0.0
#define XMAX 1.0
#define YMIN 0.0
#define YMAX 1.0

#define MAX_PSZ 10
#define TSCALE 1.0
#define VSQR 0.1

#define CUDA_CALL(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CUDA_CHK_ERR() __cudaCheckError(__FILE__, __LINE__)

/**************************************
 * void __cudaSafeCall(cudaError err, const char *file, const int line)
 * void __cudaCheckError(const char *file, const int line)
 *
 * These routines were taken from the GPU Computing SDK
 * (http://developer.nvidia.com/gpu-computing-sdk) include file "cutil.h"
 **************************************/
inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef __DEBUG

#pragma warning(push)
#pragma warning(disable : 4127) // Prevent warning on do-while(0);
  do
  {
    if (hipSuccess != err)
    {
      fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
              file, line, hipGetErrorString(err));
      exit(-1);
    }
  } while (0);
#pragma warning(pop)
#endif // __DEBUG
  return;
}

inline void __cudaCheckError(const char *file, const int line)
{
#ifdef __DEBUG
#pragma warning(push)
#pragma warning(disable : 4127) // Prevent warning on do-while(0);
  do
  {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
      fprintf(stderr, "cudaCheckError() failed at %s:%i : %s.\n",
              file, line, hipGetErrorString(err));
      exit(-1);
    }
    // More careful checking. However, this will affect performance.
    // Comment if not needed.
    /*err = cudaThreadSynchronize();
    if( cudaSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n",
               file, line, cudaGetErrorString( err ) );
      exit( -1 );
    }*/
  } while (0);
#pragma warning(pop)
#endif // __DEBUG
  return;
}

static __device__ double f(double p, double t)
{
  return -expf(-TSCALE * t) * p;
}

__global__ void EvolveKernel(double *un, double *uc, double *uo, double *pebbles, int n, double h, double t, double dt)
{
  // determine gpu thread's index into the arrays
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x > n - 1 || y > n - 1)
  {
    return;
  }
  int idx = y * n + x;

  // set edges to 0
  if (x == 0 || x == n - 1 || y == 0 || y == n - 1)
  {
    un[idx] = 0.;
  }
  else
  {
    un[idx] = 2 * uc[idx] - uo[idx] + VSQR * (dt * dt) * ((uc[idx - 1] + uc[idx + 1] + uc[idx + n] + uc[idx - n] + ((uc[idx - n - 1] + uc[idx - n + 1] + uc[idx + n - 1] + uc[idx + n + 1]) / 4) - 5 * uc[idx]) / (h * h) + f(pebbles[idx], t));
  }
}

static int tpdt(double *t, double dt, double tf)
{
  if ((*t) + dt > tf)
    return 0;
  (*t) = (*t) + dt;
  return 1;
}

void run_gpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time, int nthreads)
{
  hipEvent_t kstart, kstop;
  float ktime;

  /* HW2: Define your local variables here */

  /* device arrays and timing vars*/
  double *un_d, *uc_d, *uo_d, *pebbles_d, t, dt;

  t = 0.;
  dt = h / 2.;
  /* number of blocks to launch with */
  int blocks = n / nthreads;

  dim3 dimblocks(blocks, blocks);
  dim3 dimgrid(nthreads, nthreads);

  /* Set up device timers */
  CUDA_CALL(hipSetDevice(0));
  CUDA_CALL(hipEventCreate(&kstart));
  CUDA_CALL(hipEventCreate(&kstop));

  /* HW2: Add CUDA kernel call preperation code here */
  /* allocate device arrays */
  CUDA_CALL(hipMalloc(&un_d, sizeof(double) * n * n));
  CUDA_CALL(hipMalloc(&uc_d, sizeof(double) * n * n));
  CUDA_CALL(hipMalloc(&uo_d, sizeof(double) * n * n));
  CUDA_CALL(hipMalloc(&pebbles_d, sizeof(double) * n * n));

  /* Start GPU computation timer */
  CUDA_CALL(hipEventRecord(kstart, 0));

  /* copy in initial state into device arrays */
  CUDA_CALL(hipMemcpy(uo_d, u0, sizeof(double) * n * n, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(uc_d, u1, sizeof(double) * n * n, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(pebbles_d, pebbles, sizeof(double) * n * n, hipMemcpyHostToDevice));

  /* HW2: Add main lake simulation loop here */

  while (1)
  {
    EvolveKernel<<<dimblocks, dimgrid>>>(un_d, uc_d, uo_d, pebbles_d, n, h, t, dt);

    CUDA_CALL(hipMemcpy(uo_d, uc_d, sizeof(double) * n * n, hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(uc_d, un_d, sizeof(double) * n * n, hipMemcpyDeviceToDevice));

    if (!tpdt(&t, dt, end_time))
      break;
  }

  /* copy out un device array after last timestep */
  CUDA_CALL(hipMemcpy(u, un_d, sizeof(double) * n * n, hipMemcpyDeviceToHost));

  /* Stop GPU computation timer */
  CUDA_CALL(hipEventRecord(kstop, 0));
  CUDA_CALL(hipEventSynchronize(kstop));
  CUDA_CALL(hipEventElapsedTime(&ktime, kstart, kstop));
  printf("GPU computation: %f msec\n", ktime);

  /* HW2: Add post CUDA kernel call processing and cleanup here */
  CUDA_CALL(hipFree(un_d));
  CUDA_CALL(hipFree(uc_d));
  CUDA_CALL(hipFree(uo_d));
  CUDA_CALL(hipFree(pebbles_d));

  /* timer cleanup */
  CUDA_CALL(hipEventDestroy(kstart));
  CUDA_CALL(hipEventDestroy(kstop));
}
